#include "hip/hip_runtime.h"
/*
* Copyright 2024 Digipen.  All rights reserved.
*
* Please refer to the end user license associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms
* is strictly prohibited.
*
*/
#include <hip/hip_runtime_api.h>
#include "helper.h"

/**************
	The steps for the algorithm:
	1) Declare shared memory array for N elements with the size of TILE_WIDTH_RATIO_K � TILE_WIDTH_N
	2) Declare output array variable for P elements with the size of TILE_WIDTH_N and initialize the output array variable
	3) Loop over the input tiles (the number of iterations = (k - 1) / TILE_WIDTH_RATIO_K + 1, where k is the number of the columns of matrix M.)
	   a) Load the tile of N (size = TILE_WIDTH_RATIO_K � TILE_WIDTH_N) into shared memory.
	   Note: one block has TILE_WIDTH_M threads, each loading one N element into shared memory.
	   b) Loop over elements inside the tile of N (the number of iteration = TILE_WIDTH_RATIO_K).
	      i. Load tile of matrix M into register (i.e. each thread load one M element into the local variable in this iteration)
		 ii. Loop over and update the output elements in the output array variable for P elements assigned to this thread.
			 Note: output array variable are local variables. They accumulate the partial results. In this innerloop, the number of iteration is TILE_WIDTH_N
	4) Store the output array variable to P elements (each thread stores TILE_WIDTH_N P elements and one block outputs TILE_WIDTH_N � TILE_WIDTH_M P elements).
****************************/

//P and M column-major, N row-major
__global__ 
void matrixMultiply(FLOAT_TYPE *P,       //<! [out] and mxn matrix
					FLOAT_TYPE const *M, //<! [in] an mxk matrix
					FLOAT_TYPE const *N, //<! [in] an kxn matrix
					int const m, int const n, int const k)
{
	// Shared memory for tiling input N array
	__shared__ FLOAT_TYPE N_s[TILE_WIDTH_RATIO_K][TILE_WIDTH_N]; // 1)
	FLOAT_TYPE P_reg[TILE_WIDTH_N] = { 0 };	// 2) 

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index to access the shared memory, using threadIdx.x
	int const Ns_col = tx % TILE_WIDTH_N;
	int const Ns_row = tx / TILE_WIDTH_N;

	//your code here
	int const K_IT = ((k - 1) / TILE_WIDTH_RATIO_K) + 1;
	for (int start_k = 0; start_k < K_IT; ++start_k)
	{
		int N_col = Ns_col * start_k;
		int N_row = Ns_row * start_k;

		//int N_col = tx % TILE_WIDTH_RATIO_K + bx;
		//int N_row = tx / TILE_WIDTH_RATIO_K + bx;
		//if ((some index < k) && (some index < n))
		//{
		//	//N_s[Ns_row][Ns_col] = N[];
		//}
		__syncthreads();
	}
}

void matrixMultiplyGPU(FLOAT_TYPE* P,
	FLOAT_TYPE* M,
	FLOAT_TYPE* N,
	int numMRows,
	int numNColumns,
	int numMColumns)
{
	//@@ Initialize the grid and block dimensions here

	dim3 dimGrid((numMRows - 1) / TILE_WIDTH_M + 1, (numNColumns - 1) / TILE_WIDTH_N + 1);
	dim3 dimBlock(TILE_WIDTH_M, 1);
	matrixMultiply<<<dimGrid, dimBlock>>>(P, M, N, numMRows, numNColumns, numMColumns);

	getLastCudaError("matrixMultiply failed\n");
	hipDeviceSynchronize();
}
