#include "hip/hip_runtime.h"
/*
* Copyright 2022 Digipen.  All rights reserved.
*
* Please refer to the end user license associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms
* is strictly prohibited.
*/

#include <hip/hip_runtime_api.h>
////////////////////////////////////////////////////////////////////

#define BLOCK_SIZE 32
typedef unsigned int uint;
__global__ void heatDistrCalc(float* in, float* out, uint nRowPoints)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if x and y is within the grid boundaries
	if (nRowPoints <= x || nRowPoints <= y)
		return;

	// Calculate the index of the current point in the array
	uint idx = y * nRowPoints + x;
	uint interior = nRowPoints - 1;
	if (0 < x && interior > x && 0 < y && interior > y)
	{
		out[index] = (
						in[idx - 1] +
						in[idx + 1] +
						in[idx - nRowPoints] +
						in[idx + nRowPoints]
					 ) / 4.0f;
	}
}


__global__ void heatDistrUpdate(float* in, float* out, uint nRowPoints)
{

}

extern "C" void heatDistrGPU(
	float* d_DataIn,
	float* d_DataOut,
	uint nRowPoints,
	uint nIter
)
{
	dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 DimGrid2(ceil(((float)nRowPoints) / BLOCK_SIZE), ceil(((float)nRowPoints) / BLOCK_SIZE), 1);

	for (uint k = 0; k < nIter; k++) {
		//call heatDistrCalc
		getLastCudaError("heatDistrCalc failed\n");
		//synchronize
		//call heatDistrUpdate
		getLastCudaError("heatDistrUpdate failed\n");
		//synchronize
	}
}
